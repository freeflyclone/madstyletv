#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "xutils.h"

// this is a CUDA kernel that adds "a" to "b" with results in "c"
__global__ void addKernel(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

hipError_t LoadKernel(int *dev_a, int *dev_b, int *dev_c, int n) {
	hipError_t cudaStatus;

	// Launch a kernel on the GPU with one thread for each element.
	addKernel <<<1, n>>> (dev_c, dev_a, dev_b);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		xprintf("addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	return hipSuccess;
Error:
	xprintf("%s: Nope.\n", __FUNCTION__);
	return cudaStatus;
}